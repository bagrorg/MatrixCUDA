#include "hip/hip_runtime.h"
﻿#include "kernel.cuh"

__global__ void addKernel(int* dest, const int* src1, const int* src2, size_t n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
        dest[i] = src1[i] + src2[i];
}

__host__ void add(int* dest, const int* src1, const int* src2, size_t arraySize) {
        int *device_a, *device_b, *device_c;

        hipMalloc((void **) &device_a, arraySize * sizeof(int));
        hipMalloc((void **) &device_b, arraySize * sizeof(int));
        hipMalloc((void **) &device_c, arraySize * sizeof(int));
    
    
        hipMemcpy(device_a, src1, arraySize * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(device_b, src2, arraySize * sizeof(int), hipMemcpyHostToDevice);
    
        addKernel<<<(arraySize + 127) / 128, 128>>>(device_c, device_a, device_b, arraySize);
    
        hipMemcpy(dest, device_c, arraySize * sizeof(int), hipMemcpyDeviceToHost);
 
        hipFree(device_a);
        hipFree(device_b);
        hipFree(device_c);
}
