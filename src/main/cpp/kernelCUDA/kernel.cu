#include "hip/hip_runtime.h"
﻿#include "kernel.cuh"

__global__ void addKernel(int* dest, const int* src1, const int* src2, size_t n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
        dest[i] = src1[i] + src2[i];
}

__host__ void add(int* dest, const int* src1, const int* src2, size_t arraySize) {
        int *device_a, *device_b, *device_c;

        hipMalloc((void **) &device_a, arraySize * sizeof(int));
        hipMalloc((void **) &device_b, arraySize * sizeof(int));
        hipMalloc((void **) &device_c, arraySize * sizeof(int));
    
    
        hipMemcpy(device_a, src1, arraySize * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(device_b, src2, arraySize * sizeof(int), hipMemcpyHostToDevice);
    
        addKernel<<<(arraySize + WORK_GROUP_SIZE - 1) / WORK_GROUP_SIZE, WORK_GROUP_SIZE>>>(device_c, device_a, device_b, arraySize);
    
        hipMemcpy(dest, device_c, arraySize * sizeof(int), hipMemcpyDeviceToHost);
 
        hipFree(device_a);
        hipFree(device_b);
        hipFree(device_c);
}

__global__ void sumKernel(long int* dest, const int* src, size_t arraySize) {
        int globInd = blockIdx.x * blockDim.x + threadIdx.x;
        int localInd = threadIdx.x;
        __local__ int* localPart[WORK_GROUP_SIZE];

        if (globInd >= n) localPart[localInd] = 0;
        else localPart[localInd] = src[globInd];

        for (unsigned int length = WORK_GROUP_SIZE / 2; len > 0; len /= 2) {
            if (localInd < len) localPart[localInd] = localPart[localInd] + localPart[localInd + len];
            if (len > WARP_SIZE)  __syncthreads();
            else if (localInd >= len) return;
        }

        if (localInd == 0) *dest += localPart[0];
}

__host__ void sum(long int* dest, const int* src, size_t arraySize) {
        int *device_arr;
        long int *res;
        hipMalloc((void **) &device_arr, arraySize * sizeof(int));
        hipMalloc((void **) &res, sizeof(long int));

        hipMemcpy(device_arr, src, arraySize * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(res, dest, sizeof(long int), hipMemcpyHostToDevice);

        sumKernel<<<(arraySize + WORK_GROUP_SIZE - 1) / WORK_GROUP_SIZE, WORK_GROUP_SIZE>>>(res, device_arr, arraySize);

        hipMemcpy(dest, res, sizeof(long int), hipMemcpyDeviceToHost);

        hipFree(device_arr);
        hipFree(res);
}